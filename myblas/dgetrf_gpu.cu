#include "hip/hip_runtime.h"
/**
 *
 * @file dgetrf_gpu.cu
 *
 * @copyright 2019-2021 Bordeaux INP, CNRS (LaBRI UMR 5800), Inria,
 *                      Univ. Bordeaux. All rights reserved.
 *
 * @brief Template for the initial sequential GETRF function
 *
 * This file is a template that you can copy/paste as many times as
 * you like to create new versions of the GETRF implementation.
 *
 * To do that, replace all occurence of the TEMPLATE keyword by the shortname
 * you like, and save the file under the name dgetrf_TEMPLATE.c
 * Then, add the file to the CMakeLists.txt GETRF list, compile and enjoy.
 *
 * @version 0.2.0
 * @author YOURSELF
 * @date 2021-09-30
 *
 */
#include "myblas.h"
#include <hip/hip_runtime.h>

#define REG_NB_ELEMENTS 4

__global__ void dgetrf_gpu_kernel(int M, int N, double *A, int lda) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < M) {
        int k;
        __m256d reg_minus1 = _mm256_set1_pd(-1.0);

        for (k = 0; k < min(M, N); k++) {
            if (tid == k) {
                // Division
                A[lda * k + tid] /= A[lda * k + k];
            }
            __syncthreads();

            if (tid > k && tid < M) {
                // Calculs vectorisés
                __m256d reg_A = _mm256_loadu_pd(&A[lda * k + tid]);
                __m256d reg_KK = _mm256_set1_pd(A[lda * k + k]);
                reg_A = _mm256_div_pd(reg_A, reg_KK);
                _mm256_storeu_pd(&A[lda * k + tid], reg_A);
            }
            __syncthreads();

            if (tid > k && tid < M) {
                // Calculs vectorisés
                __m256d reg_AK = _mm256_loadu_pd(&A[lda * k + tid]);
                __m256d reg_AKN = _mm256_set1_pd(A[lda * tid + k]);
                __m256d reg_AN = _mm256_loadu_pd(&A[lda * tid + k]);
                reg_AK = _mm256_fmadd_pd(reg_minus1, reg_AK, _mm256_mul_pd(reg_AK, reg_AKN));
                reg_AN = _mm256_sub_pd(reg_AN, _mm256_mul_pd(reg_AK, reg_AN));
                _mm256_storeu_pd(&A[lda * tid + k], reg_AN);
            }
            __syncthreads();
        }
    }
}

int dgetrf_gpu( CBLAS_LAYOUT layout, CBLAS_TRANSPOSE transA,
                    CBLAS_TRANSPOSE transB, const int M, const int N,
                    const int K, const double alpha, const double *A,
                    const int lda, const double *B, const int ldb,
                    const double beta, double *C, const int ldc )
{
    /* Here is where you put your own code */

    double *d_A;

    // Allouer de la mémoire sur le GPU
    hipMalloc((void**)&d_A, M * lda * sizeof(double));

    // Copier les données depuis le CPU vers le GPU
    hipMemcpy(d_A, A, M * lda * sizeof(double), hipMemcpyHostToDevice);

    // Définir la grille et la taille des blocs pour les threads du GPU
    dim3 gridSize((M + REG_NB_ELEMENTS - 1) / REG_NB_ELEMENTS, 1, 1);
    dim3 blockSize(REG_NB_ELEMENTS, 1, 1);

    // Appeler le noyau GPU
    dgetrf_gpu_kernel<<<gridSize, blockSize>>>(M, N, d_A, lda);

    // Attendre la fin des calculs du GPU
    hipDeviceSynchronize();

    // Copier les résultats depuis le GPU vers le CPU
    hipMemcpy(A, d_A, M * lda * sizeof(double), hipMemcpyDeviceToHost);

    // Libérer la mémoire allouée sur le GPU
    hipFree(d_A);

    return ALGONUM_SUCCESS;
}

/* To make sure we use the right prototype */
static dgetrf_fct_t valid_dgetrf_gpu __attribute__ ((unused)) = dgetrf_gpu;

/* Declare the variable that will store the information about this version */
fct_list_t fct_dgetrf_gpu;

/**
 * @brief Registration function
 */
void dgetrf_gpu_init( void ) __attribute__( ( constructor ) );
void
dgetrf_gpu_init( void )
{
    fct_dgetrf_TEMPLATE.tiled  = 0;
    fct_dgetrf_TEMPLATE.starpu = 0;
    fct_dgetrf_TEMPLATE.name   = "gpu";
    fct_dgetrf_TEMPLATE.helper = "CUDA version of dgetrf";
    fct_dgetrf_TEMPLATE.fctptr = dgetrf_gpu;
    fct_dgetrf_TEMPLATE.next   = NULL;

    register_fct( &fct_dgetrf_TEMPLATE, ALGO_GETRF );
}
